
#include <hip/hip_runtime.h>
#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
	  i < (n);                                       \
      i += blockDim.x * gridDim.x)
      
#define INDEX(b,c,h,w,channels,height,width) ((b * channels + c) * height + h) * width+ w


extern "C" __global__ void IRNNBackward(
    float* 				grad_input,
    
    float*      		grad_weight_up_map,
    float*              grad_weight_right_map,
    float*              grad_weight_down_map,
    float*              grad_weight_left_map,

    float*    			grad_bias_up_map,
    float*   			grad_bias_right_map,
    float*    			grad_bias_down_map,
    float*    			grad_bias_left_map,

    const float*		weight_up,
    const float*        weight_right,
    const float*        weight_down,
    const float*        weight_left,

    const float*		grad_output_up, 
    const float*        grad_output_right,
    const float*        grad_output_down,
    const float*        grad_output_left,

    const float*  		output_up,
    const float*        output_right,
    const float*        output_down,
    const float*        output_left,

	const int 			channels, 
	const int 			height, 
	const int			width,
    const int           n) {

    CUDA_KERNEL_LOOP(index,n){

        int w = index % width;
        int h = index / width % height;
        int c = index / width / height % channels;
        int b = index / width / height / channels;

        float diff_left  = 0;
        float diff_right = 0;
        float diff_up    = 0;
        float diff_down  = 0;

        //left 
       
        for (int i = 0; i<=w; i++)
        {   
            diff_left *= weight_left[c];
            diff_left += grad_output_left[INDEX(b, c, h, i, channels, height, width)];
            diff_left *= (output_left[INDEX(b, c, h, i, channels, height, width)]<=0)? 0 : 1;
        }
        

        float temp = grad_output_left[INDEX(b, c, h, 0, channels, height, width)];
        for (int i = 1; i < w +1 ; i++)
        {
            temp = (output_left[INDEX(b, c, h, i-1, channels, height, width)] >0?1:0) * temp * weight_left[c] + grad_output_left[INDEX(b, c, h, i, channels, height, width)];
        }

        if (w != width - 1){
            grad_weight_left_map[index] = temp * output_left[INDEX(b, c, h, w+1, channels, height, width)] * (output_left[index] > 0? 1:0);
            grad_bias_left_map[index] = diff_left;
        }

        // right 

        for (int i = width -1; i>=w; i--)
        {   
            diff_right *= weight_right[c];
            diff_right += grad_output_right[INDEX(b, c, h, i, channels, height, width)];
            diff_right *= (output_right[INDEX(b, c, h, i, channels, height, width)]<=0)? 0 : 1;
        }
        

        temp = grad_output_right[INDEX(b, c, h, width-1, channels, height, width)];
        for (int i = width -2; i > w - 1 ; i--)
        {
            temp = (output_right[INDEX(b, c, h, i+1, channels, height, width)] >0?1:0) * temp * weight_right[c] + grad_output_right[INDEX(b, c, h, i, channels, height, width)];
        }

        if (w != 0){
            grad_weight_right_map[index] = temp * output_right[INDEX(b, c, h, w-1, channels, height, width)] * (output_right[index] > 0? 1:0);
            grad_bias_right_map[index] = diff_right;
        }

        // up

        
        for (int i = 0; i<=h; i++)
        {   
            diff_up *= weight_up[c];
            diff_up += grad_output_up[INDEX(b, c, i, w, channels, height, width)];
            diff_up *= (output_up[INDEX(b, c, i, w, channels, height, width)]<=0)? 0 : 1;
        }
       

        temp = grad_output_up[INDEX(b, c, 0, w, channels, height, width)];
        for (int i = 1; i < h +1 ; i++)
        {
            temp = (output_up[INDEX(b, c, i-1, w, channels, height, width)] >0?1:0) * temp * weight_up[c] + grad_output_up[INDEX(b, c, i, w, channels, height, width)];
        }

        if (h != height - 1){
            grad_weight_up_map[index] = temp * output_up[INDEX(b, c, h+1, w, channels, height, width)] * (output_up[index] > 0? 1:0);
            grad_bias_up_map[index] = diff_up;
        }

        // down

        for (int i = height -1; i>=h; i--)
        {   
            diff_down *= weight_down[c];
            diff_down += grad_output_down[INDEX(b, c, i, w, channels, height, width)];
            diff_down *= (output_down[INDEX(b, c, i, w, channels, height, width)]<=0)? 0 : 1;
        }
        

        temp = grad_output_down[INDEX(b, c, height-1, w, channels, height, width)];
        for (int i = height -2; i > h - 1 ; i--)
        {
            temp = (output_down[INDEX(b, c, i+1, w, channels, height, width)] >0?1:0) * temp * weight_down[c] + grad_output_down[INDEX(b, c, i, w, channels, height, width)];
        }

        if (h != 0){
            grad_weight_down_map[index] = temp * output_down[INDEX(b, c, h-1, w, channels, height, width)] * (output_down[index] > 0? 1:0);
            grad_bias_down_map[index] = diff_down;
        }
        grad_input[index] = diff_down + diff_left + diff_right + diff_up;
    }
}
